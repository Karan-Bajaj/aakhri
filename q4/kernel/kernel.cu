
#include <hip/hip_runtime.h>
#include <stdio.h>

// Use this as the random seed in the pseudorandom function (see instructions)
#define SEED 20210418

// Pseudo-random number generator. This provides consistent reproducible results so that
// you can test your program.
__device__ float pseudorandom( uint x ) {
    uint value = x;
    value = (value ^ 61) ^ (value>>16);
    value *= 9;
    value ^= value << 4;
    value *= 0x27d4eb2d;
    value ^= value >> 15;
    return (float) value / (float) INT_MAX;
}
